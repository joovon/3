
#include <hip/hip_runtime.h>
// dst[i] = fac * src[i]
extern "C" __global__ void
mul(float* __restrict__  dst, float* __restrict__  src, float fac, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if(i < N) {
        dst[i] = fac * src[i];
    }
}

